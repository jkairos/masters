
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 100000
#define THREAD_PER_BLOCK 1

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

__global__ void add(int *a, int *b, int *c) {
	int tid = blockIdx.x; // handle the data at this index
	if (tid < N) {
		c[tid] = a[tid] + b[tid];
	}
}

void vector_add(int *a, int *b, int *c){
	for(int i=0; i <N; i++){
		c[i] = a[i]+b[i];
	}
}

void displayResults(int *a, int *b, int *c){
	// display the results
	for (int i = 0; i < N; i++) {
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}
}

void sumVec(int *a, int *b, int *c, int start, int end){
	if(end < start || start > end){
		return;
	}

//	int idx= (start + (end-start))/2;
	c[end]=a[end]+b[end];
//	c[idx]=a[idx]+b[idx];
	c[start]=a[start]+b[start];

	sumVec(a,b,c,start+1,end-1);
//	sumVec(a,b,c,idx+1,end-1);
}

void cpuImplementation(){
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	//allocate and initialize host cpu memory
	int a[N];
	int b[N];
	int c[N];

	//fill the arrays 'h_a' and 'h_b' on the CPU
	for(int i=0; i< N; i++){
		a[i]=-i;
		b[i]=i*i;
		c[i]=0;
	}

	hipEventRecord(start);
	//vector_add(a,b,c);
	sumVec(a,b,c,0,N-1);
	hipEventRecord(stop);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
//	displayResults(a,b,c);
	printf("Elapsed Time in CPU %fms\n", milliseconds);
}

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
int main(void) {
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;
	hipEvent_t start, stop;
	int threadPerBlock=THREAD_PER_BLOCK;

	cpuImplementation();

	hipEventCreate(&start);
	hipEventCreate(&stop);

	// allocate the memory on the GPU
	CUDA_CHECK_RETURN(hipMalloc( (void**)&dev_a, N * sizeof(int) ));
	CUDA_CHECK_RETURN(hipMalloc( (void**)&dev_b, N * sizeof(int) ));
	CUDA_CHECK_RETURN(hipMalloc( (void**)&dev_c, N * sizeof(int) ));

	// fill the arrays 'a' and 'b' on the CPU
	for (int i = 0; i < N; i++) {
		a[i] = -i;
		b[i] = i * i;
	}

	// copy the arrays 'a' and 'b' to the GPU
	CUDA_CHECK_RETURN(hipMemcpy( dev_a, a, N * sizeof(int), hipMemcpyHostToDevice ));
	CUDA_CHECK_RETURN(hipMemcpy( dev_b, b, N * sizeof(int), hipMemcpyHostToDevice ));
	hipEventRecord(start);
	add<<<N, threadPerBlock>>>(dev_a, dev_b, dev_c);
	hipEventRecord(stop);
	// copy the array 'c' back from the GPU to the CPU
	CUDA_CHECK_RETURN(hipMemcpy( c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost ));

//	displayResults(a,b,c);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	printf("Elapsed Time in GPU %fms\n", milliseconds);

	// free the memory allocated on the GPU
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);


	return 0;
}
